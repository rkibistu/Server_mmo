#include "hip/hip_runtime.h"
#include "GpuCollisions.h"

__global__ void DetectBoxCollisionsKernel(rml::Vector3* minAABB, rml::Vector3* maxAABB, int* moveIds, int* ids, int size, int* collisionMarkers) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	if (moveIds[i] == 0) return; // check onyl for objects that moved

	for (int j = 0; j < size; j++) {
		if (i == j)
			continue;

		if ((minAABB[i].x <= maxAABB[j].x && maxAABB[i].x >= minAABB[j].x) &&
			(minAABB[i].y <= maxAABB[j].y && maxAABB[i].y >= minAABB[j].y) &&
			(minAABB[i].z <= maxAABB[j].z && maxAABB[i].z >= minAABB[j].z)) {

			//int x = ids[i];

			collisionMarkers[ids[i]] = 1;
		}
	}
}

hipError_t GPU_DetectBoxCollisions(rml::Vector3* minAABB, rml::Vector3* maxAABB, int* moveIds, int* ids, int size, int* collisionMarkers, int collisionMarkersSize) {

	rml::Vector3* devMinAABB;
	rml::Vector3* devMaxAABB;
	int* devCollisionMarkers;
	int* devMoveIds;
	int* devIds;

	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devMinAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 0 !");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMaxAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 1!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devCollisionMarkers, collisionMarkersSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 2!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMoveIds, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 3!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devIds, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed 4!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devMinAABB, minAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy minaabb failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMaxAABB, maxAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy maxaabb failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMoveIds, moveIds, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy maxaabb failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(devIds, ids, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy maxaabb failed!\n");
		goto Error;
	}
	hipMemset(devCollisionMarkers, 0, collisionMarkersSize * sizeof(int));


	//CallTheKernel
	// Launch a kernel on the GPU with one thread for each element.
	DetectBoxCollisionsKernel << <numBlocks, threadsPerBlock >> > (devMinAABB, devMaxAABB, devMoveIds, devIds, size, devCollisionMarkers);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(collisionMarkers, devCollisionMarkers, collisionMarkersSize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy genC failed!\n");
		goto Error;
	}


Error:
	hipFree(devMinAABB);
	hipFree(devMaxAABB);
	hipFree(devCollisionMarkers);
	hipFree(devMoveIds);
	hipFree(devIds);

	return cudaStatus;
}
