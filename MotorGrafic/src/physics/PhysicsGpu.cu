#include "hip/hip_runtime.h"
#include "PhysicsGpu.h"
#include "PhysicsController.h"

#include <iostream>
#include <stdio.h>

__global__ void UpdateBoxesKernel(rml::Vector3* position, rml::Vector3* velocity, rml::Vector3* minAABB, rml::Vector3* maxAABB, float gravity, float dragCoefficient, rml::Vector3 boxMinBound, rml::Vector3 boxMaxBound, float deltaTime) {



	//gravity
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	velocity[i].y -= gravity * deltaTime;

	//friction
	//float vectorLength = sqrt(velocity[i].x * velocity[i].x + velocity[i].y * velocity[i].y + velocity[i].z * velocity[i].z);
	//if (vectorLength > 0.0f) { // Only apply drag if the object is moving
	//	// Normalize the velocity vector to get the direction
	//	// rml::Vector3 velocityDirection = velocity[i];
	//	// velocityDirection = velocityDirection.Normalize();
	//	float xNorm = velocity[i].x * (1.0f / vectorLength);
	//	float yNorm = velocity[i].y * (1.0f / vectorLength);
	//	float zNorm = velocity[i].z * (1.0f / vectorLength);

	//	// Calculate the drag force (opposing direction of motion)
	//	// rml::Vector3 dragForce = -velocityDirection * dragCoefficient * speed;

	//	// Apply the drag force to the object's velocity
	//	//velocity[i] += dragForce;
	//	velocity[i].x += -xNorm * dragCoefficient * vectorLength;
	//	velocity[i].y += -yNorm * dragCoefficient * vectorLength;
	//	velocity[i].z += -zNorm * dragCoefficient * vectorLength;
	//}

	////update pos
	position[i].x += velocity[i].x * deltaTime;
	position[i].y += velocity[i].y * deltaTime;
	position[i].z += velocity[i].z * deltaTime;

	//wall collisions
	float _wallRestitution = 0.1f;
	float minBoundX = minAABB[i].x + position[i].x;
	float minBoundY = minAABB[i].y + position[i].y;
	float minBoundZ = minAABB[i].z + position[i].z;

	float maxBoundX = maxAABB[i].x + position[i].x;
	float maxBoundY = maxAABB[i].y + position[i].y;
	float maxBoundZ = maxAABB[i].z + position[i].z;

	if (minBoundX < boxMinBound.x) {
		position[i].x = boxMinBound.x + (position[i].x - minBoundX);
		velocity[i].x *= -_wallRestitution;
	}
	else if (maxBoundX > boxMaxBound.x) {
		position[i].x = boxMaxBound.x - (maxBoundX - position[i].x);
		velocity[i].x *= -_wallRestitution;
	}

	if (minBoundY < boxMinBound.y) {
		position[i].y = boxMinBound.y + (position[i].y - minBoundY);
		velocity[i].y *= -_wallRestitution;
	}
	else if (maxBoundY > boxMaxBound.y) {
		position[i].y = boxMaxBound.y - (maxBoundY - position[i].y);
		velocity[i].y *= -_wallRestitution;
	}

	if (minBoundZ < boxMinBound.z) {
		position[i].z = boxMinBound.z + (position[i].z - minBoundZ);
		velocity[i].z *= -_wallRestitution;
	}
	else if (maxBoundZ > boxMaxBound.z) {
		position[i].z = boxMaxBound.z - (maxBoundZ - position[i].z);
		velocity[i].z *= -_wallRestitution;
	}
}

__global__ void ApplyGravityKernel(rml::Vector3* velocity, int size, float gravity, float deltaTime) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	velocity[i].y -= gravity * deltaTime;
}
__global__ void ApplyFrictionKernel(rml::Vector3* velocity, int size, float drag, float deltaTime) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float vectorLength = sqrt(velocity[i].x * velocity[i].x + velocity[i].y * velocity[i].y + velocity[i].z * velocity[i].z);
	if (vectorLength > 0.0f) { // Only apply drag if the object is moving
		// Normalize the velocity vector to get the direction
		// rml::Vector3 velocityDirection = velocity[i];
		// velocityDirection = velocityDirection.Normalize();
		float xNorm = velocity[i].x * (1.0f / vectorLength);
		float yNorm = velocity[i].y * (1.0f / vectorLength);
		float zNorm = velocity[i].z * (1.0f / vectorLength);

		// Calculate the drag force (opposing direction of motion)
		// rml::Vector3 dragForce = -velocityDirection * dragCoefficient * speed;

		// Apply the drag force to the object's velocity
		//velocity[i] += dragForce;
		velocity[i].x += -xNorm * drag * vectorLength;
		velocity[i].y += -yNorm * drag * vectorLength;
		velocity[i].z += -zNorm * drag * vectorLength;
	}
}
__global__ void UpdateApplyingGravityAndFrictionKernel(rml::Vector3* position, rml::Vector3* velocity, int size, float gravity, float drag, float deltaTime) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	velocity[i].y -= gravity * deltaTime;

	float vectorLength = sqrt(velocity[i].x * velocity[i].x + velocity[i].y * velocity[i].y + velocity[i].z * velocity[i].z);
	if (vectorLength > 0.0f) { // Only apply drag if the object is moving
		// Normalize the velocity vector to get the direction
		// rml::Vector3 velocityDirection = velocity[i];
		// velocityDirection = velocityDirection.Normalize();
		float xNorm = velocity[i].x * (1.0f / vectorLength);
		float yNorm = velocity[i].y * (1.0f / vectorLength);
		float zNorm = velocity[i].z * (1.0f / vectorLength);

		// Calculate the drag force (opposing direction of motion)
		// rml::Vector3 dragForce = -velocityDirection * dragCoefficient * speed;

		// Apply the drag force to the object's velocity
		//velocity[i] += dragForce;
		velocity[i].x += -xNorm * drag * vectorLength;
		velocity[i].y += -yNorm * drag * vectorLength;
		velocity[i].z += -zNorm * drag * vectorLength;
	}

	// If you comment the line that udpates Y position and update it from the CPU, it works good.
	// If you update it here, all the objects go to left? How ? why ?
	position[i].x = position[i].x + velocity[i].x * deltaTime;
	position[i].y = position[i].y + velocity[i].y * deltaTime;
	position[i].z = position[i].z + velocity[i].z * deltaTime;
}

__global__ void HandleBoxWallCollisionsKernel(rml::Vector3* position, rml::Vector3* velocity, rml::Vector3* minAABB, rml::Vector3* maxAABB, rml::Vector3 minBoxBound, rml::Vector3 maxBoxBound, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float _wallRestitution = 0.1f;
	float minBoundX = minAABB[i].x;//+ position[i].x;
	float minBoundY = minAABB[i].y;//+ position[i].y;
	float minBoundZ = minAABB[i].z;//+ position[i].z;

	float maxBoundX = maxAABB[i].x;//+ position[i].x;
	float maxBoundY = maxAABB[i].y;//+ position[i].y;
	float maxBoundZ = maxAABB[i].z;// +position[i].z;

	if (minBoundX < minBoxBound.x) {
		position[i].x = minBoxBound.x + (position[i].x - minBoundX);
		velocity[i].x *= -_wallRestitution;
	}
	else if (maxBoundX > maxBoxBound.x) {
		position[i].x = maxBoxBound.x - (maxBoundX - position[i].x);
		velocity[i].x *= -_wallRestitution;
	}

	if (minBoundY < minBoxBound.y) {
		position[i].y = minBoxBound.y + (position[i].y - minBoundY);
		velocity[i].y *= -_wallRestitution;
	}
	else if (maxBoundY > maxBoxBound.y) {
		position[i].y = maxBoxBound.y - (maxBoundY - position[i].y);
		velocity[i].y *= -_wallRestitution;
	}

	if (minBoundZ < minBoxBound.z) {
		position[i].z = minBoxBound.z + (position[i].z - minBoundZ);
		velocity[i].z *= -_wallRestitution;
	}
	else if (maxBoundZ > maxBoxBound.z) {
		position[i].z = maxBoxBound.z - (maxBoundZ - position[i].z);
		velocity[i].z *= -_wallRestitution;
	}
}
__global__ void HandleSphereWallCollisionsKernel(rml::Vector3* position, rml::Vector3* velocity, float* radius, rml::Vector3 minBoxBound, rml::Vector3 maxBoxBound, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	float _wallRestitution = 0.5f;
	if (position[i].x - radius[i] < minBoxBound.x) {
		position[i].x = minBoxBound.x + radius[i];
		velocity[i].x *= -_wallRestitution;
	}
	else if (position[i].x + radius[i] > maxBoxBound.x) {
		position[i].x = maxBoxBound.x - radius[i];
		velocity[i].x *= -_wallRestitution;
	}
	if (position[i].y - radius[i] < minBoxBound.y) {
		position[i].y = minBoxBound.y + radius[i];
		velocity[i].y *= -_wallRestitution;
	}
	else if (position[i].y + radius[i] > maxBoxBound.y) {
		position[i].y = maxBoxBound.y - radius[i];
		velocity[i].y *= -_wallRestitution;
	}
	if (position[i].z - radius[i] < minBoxBound.z) {
		position[i].z = minBoxBound.z + radius[i];
		velocity[i].z *= -_wallRestitution;
	}
	else if (position[i].z + radius[i] > maxBoxBound.z) {
		position[i].z = maxBoxBound.z - radius[i];
		velocity[i].z *= -_wallRestitution;
	}
}

__global__ void HandleBoxCollisionsKernel(rml::Vector3* position, rml::Vector3* velocity, rml::Vector3* minAABB, rml::Vector3* maxAABB, int size, float collisionRestitution, int i) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;


	if (i <= j) return;

	// Check for overlap between the bounding volumes (AABBs)
	if ((minAABB[i].x <= maxAABB[j].x && maxAABB[i].x >= minAABB[j].x) &&
		(minAABB[i].y <= maxAABB[j].y && maxAABB[i].y >= minAABB[j].y) &&
		(minAABB[i].z <= maxAABB[j].z && maxAABB[i].z >= minAABB[j].z)) {
		// Determine the penetration depth along each axis
		float penetrationX;
		float penetrationY;
		float penetrationZ;


		//penetration.x = std::min(maxAABB[i].x - minAABB[j].x, maxAABB[j].x - minAABB[i].x);
		//penetration.y = std::min(maxAABB[i].y - minAABB[j].y, maxAABB[j].y - minAABB[i].y);
		//penetration.z = std::min(maxAABB[i].z - minAABB[j].z, maxAABB[j].z - minAABB[i].z);
		penetrationX = (maxAABB[i].x - minAABB[j].x < maxAABB[j].x - minAABB[i].x) ?
			(maxAABB[i].x - minAABB[j].x) :
			(maxAABB[j].x - minAABB[i].x);

		penetrationY = (maxAABB[i].y - minAABB[j].y < maxAABB[j].y - minAABB[i].y) ?
			(maxAABB[i].y - minAABB[j].y) :
			(maxAABB[j].y - minAABB[i].y);

		penetrationZ = (maxAABB[i].z - minAABB[j].z < maxAABB[j].z - minAABB[i].z) ?
			(maxAABB[i].z - minAABB[j].z) :
			(maxAABB[j].z - minAABB[i].z);

		// Threshold to avoid jitter on small penetrations
/*		const float penetrationThreshold = 0.01f;
		if (penetration.x < penetrationThreshold && penetration.y < penetrationThreshold && penetration.z < penetrationThreshold) {
			return;
		}*/

		// Find the axis of least penetration to separate objects along that axis
		if (penetrationX < penetrationY && penetrationX < penetrationZ) {
			if (position[i].x < position[j].x) {
				position[i].x -= penetrationX / 2;
				position[j].x += penetrationX / 2;
			}
			else {
				position[i].x += penetrationX / 2;
				position[j].x -= penetrationX / 2;
			}
			velocity[i].x *= -collisionRestitution;
			velocity[j].x *= -collisionRestitution;
		}
		else if (penetrationY < penetrationZ) {
			if (position[i].y < position[j].y) {
				position[i].y -= penetrationY / 2;
				position[j].y += penetrationY / 2;
			}
			else {
				position[i].y += penetrationY / 2;
				position[j].y -= penetrationY / 2;
			}
			velocity[i].y *= -collisionRestitution;
			velocity[j].y *= -collisionRestitution;
		}
		else {
			if (position[i].z < position[j].z) {
				position[i].z -= penetrationZ / 2;
				position[j].z += penetrationZ / 2;
			}
			else {
				position[i].z += penetrationZ / 2;
				position[j].z -= penetrationZ / 2;
			}
			velocity[i].z *= -collisionRestitution;
			velocity[j].z *= -collisionRestitution;
		}
	}

}

__global__ void DetectBoxCollisionsKernel(rml::Vector3* position, rml::Vector3* minAABB, rml::Vector3* maxAABB, int size, int* generateCollisonPointsCount, CollisionPoint* collisionPoints, int cpSize) {
	//current eleemnt to check versus all of them
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;


	// fiecare thread poate folosi zona asta fara sa isi faca griji de acces concurent la memorie (vectorul collisionPoints)
	// i * size -> i * size + size - 1

	//the number fo collisions detected for object i
	int countCollisions = 0;
	float penetrationX, penetrationY, penetrationZ;
	for (int j = i + 1; j < size; j++) {

		//check if object i collides with object j
		if ((minAABB[i].x <= maxAABB[j].x && maxAABB[i].x >= minAABB[j].x) &&
			(minAABB[i].y <= maxAABB[j].y && maxAABB[i].y >= minAABB[j].y) &&
			(minAABB[i].z <= maxAABB[j].z && maxAABB[i].z >= minAABB[j].z)) {

			//Calculate penetration
			penetrationX = (maxAABB[i].x - minAABB[j].x < maxAABB[j].x - minAABB[i].x) ? (maxAABB[i].x - minAABB[j].x) : (maxAABB[j].x - minAABB[i].x);
			penetrationY = (maxAABB[i].y - minAABB[j].y < maxAABB[j].y - minAABB[i].y) ? (maxAABB[i].y - minAABB[j].y) : (maxAABB[j].y - minAABB[i].y);
			penetrationZ = (maxAABB[i].z - minAABB[j].z < maxAABB[j].z - minAABB[i].z) ? (maxAABB[i].z - minAABB[j].z) : (maxAABB[j].z - minAABB[i].z);

			//Check for threshold
			if (penetrationX > 0.01 || penetrationY > 0.01 || penetrationZ > 0.01) {
				//Add to the penetration array
				collisionPoints[i * size + countCollisions].Object1 = i;
				collisionPoints[i * size + countCollisions].Object2 = j;
				collisionPoints[i * size + countCollisions].Penetration.x = penetrationX;
				collisionPoints[i * size + countCollisions].Penetration.y = penetrationY;
				collisionPoints[i * size + countCollisions].Penetration.z = penetrationZ;

				countCollisions++;
			}
		}
	}

	generateCollisonPointsCount[i] = countCollisions;

	//generateCollisonPointsCount[i] = 1;

	//collisionPoints[i * size + 0].Penetration.x = 10;
	//collisionPoints[i * size + 0].Penetration.y = 10;
	//collisionPoints[i * size + 0].Penetration.z = 10;
}
__global__ void DetectSphereCollisionsKernel(rml::Vector3* position, float* radius, int size, int* generateCollisonPointsCount, CollisionPoint* collisionPoints, int cpSize) {
	//current eleemnt to check versus all of them
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;

	int countCollisions = 0;
	float penetrationX, penetrationY, penetrationZ;
	for (int j = i + 1; j < size; j++) {

		float distanceVecX = position[j].x - position[i].x;
		float distanceVecY = position[j].y - position[i].y;
		float distanceVecZ = position[j].z - position[i].z;
		double distance = sqrt(distanceVecX * distanceVecX + distanceVecY * distanceVecY + distanceVecZ * distanceVecZ);
		double collisionDistance = radius[i] + radius[j];

		if (distance - collisionDistance < 0.01f) {
			//rml::Vector3 normal = distanceVec.Normalize();
			float xNorm = distanceVecX;
			float yNorm = distanceVecY;
			float zNorm = distanceVecZ;
			if (distance != 0) {

				float xNorm = distanceVecX * (1.0f / distance);
				float yNorm = distanceVecY * (1.0f / distance);
				float zNorm = distanceVecZ * (1.0f / distance);
			}

			double penetrationDepth = collisionDistance - distance;

			float xPenetration = xNorm * penetrationDepth;
			float yPenetration = yNorm * penetrationDepth;
			float zPenetration = zNorm * penetrationDepth;

			collisionPoints[i * size + countCollisions].Object1 = i;
			collisionPoints[i * size + countCollisions].Object2 = j;
			collisionPoints[i * size + countCollisions].Penetration.x = xPenetration;
			collisionPoints[i * size + countCollisions].Penetration.y = yPenetration;
			collisionPoints[i * size + countCollisions].Penetration.z = zPenetration;

			countCollisions++;
		}
	}
	generateCollisonPointsCount[i] = countCollisions;
}

hipError_t GPU_ApplyGravity(rml::Vector3* velocity, int size, float gravity, float deltaTime) {
	rml::Vector3* devVelocity;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	ApplyGravityKernel << <numBlocks, threadsPerBlock >> > (devVelocity, size, gravity, deltaTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devVelocity);

	return cudaStatus;
}

hipError_t GPU_ApplyFriction(rml::Vector3* velocity, int size, float drag, float deltaTime) {
	rml::Vector3* devVelocity;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	ApplyFrictionKernel << <numBlocks, threadsPerBlock >> > (devVelocity, size, drag, deltaTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devVelocity);

	return cudaStatus;
}

hipError_t GPU_UpdateApplyingGravityAndFriction(rml::Vector3* position, rml::Vector3* velocity, int size, float gravity, float drag, float deltaTime) {
	rml::Vector3* devPosition;
	rml::Vector3* devVelocity;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	UpdateApplyingGravityAndFrictionKernel << <numBlocks, threadsPerBlock >> > (devPosition, devVelocity, size, gravity, drag, deltaTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(position, devPosition, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devVelocity);
	hipFree(devPosition);

	return cudaStatus;
}

hipError_t GPU_HandleBoxWallCollisions(rml::Vector3* position, rml::Vector3* velocity, rml::Vector3* minAABB, rml::Vector3* maxAABB, rml::Vector3 minBoxBound, rml::Vector3 maxBoxBound, int size) {
	rml::Vector3* devPosition;
	rml::Vector3* devVelocity;
	rml::Vector3* devMinAABB;
	rml::Vector3* devMaxAABB;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMinAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMaxAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMinAABB, minAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMaxAABB, maxAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	HandleBoxWallCollisionsKernel << <numBlocks, threadsPerBlock >> > (devPosition, devVelocity, devMinAABB, devMaxAABB, minBoxBound, maxBoxBound, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(position, devPosition, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devVelocity);
	hipFree(devPosition);
	hipFree(devMinAABB);
	hipFree(devMaxAABB);

	return cudaStatus;
}

hipError_t GPU_HandleSphereWallCollisions(rml::Vector3* position, rml::Vector3* velocity, float* radius, rml::Vector3 minBoxBound, rml::Vector3 maxBoxBound, int size) {
	rml::Vector3* devPosition;
	rml::Vector3* devVelocity;
	float* devRadius;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devRadius, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devRadius, radius, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	// Launch a kernel on the GPU with one thread for each element.
	HandleSphereWallCollisionsKernel << <numBlocks, threadsPerBlock >> > (devPosition, devVelocity, devRadius, minBoxBound, maxBoxBound, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(position, devPosition, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devVelocity);
	hipFree(devPosition);
	hipFree(devRadius);

	return cudaStatus;
}

hipError_t GPU_DetectBoxCollisions(rml::Vector3* position, rml::Vector3* minAABB, rml::Vector3* maxAABB, int size, int* generateCollisonPointsCount, CollisionPoint* collisionPoints, int cpSize) {

	rml::Vector3* devPosition;
	rml::Vector3* devMinAABB;
	rml::Vector3* devMaxAABB;
	int* devGenerateCollisionPointsCount;
	CollisionPoint* devCollisionPoints;

	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMinAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMaxAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devGenerateCollisionPointsCount, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devCollisionPoints, cpSize * sizeof(CollisionPoint));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy pos failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMinAABB, minAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy minaabb failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMaxAABB, maxAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy maxaabb failed!\n");
		goto Error;
	}
	hipMemset(devGenerateCollisionPointsCount, 0, size * sizeof(int));
	hipMemset(devCollisionPoints, 0, cpSize * sizeof(CollisionPoint));

	//CallTheKernel
	// Launch a kernel on the GPU with one thread for each element.
	DetectBoxCollisionsKernel << <numBlocks, threadsPerBlock >> > (devPosition, devMinAABB, devMaxAABB, size, devGenerateCollisionPointsCount, devCollisionPoints, cpSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(generateCollisonPointsCount, devGenerateCollisionPointsCount, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy genC failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(collisionPoints, devCollisionPoints, cpSize * sizeof(CollisionPoint), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy cp failed!\n");
		goto Error;
	}

Error:
	hipFree(devPosition);
	hipFree(devMinAABB);
	hipFree(devMaxAABB);
	hipFree(devGenerateCollisionPointsCount);
	hipFree(devCollisionPoints);

	return cudaStatus;
}

hipError_t GPU_DetectSphereCollisions(rml::Vector3* position, float* radius, int size, int* generateCollisonPointsCount, CollisionPoint* collisionPoints, int cpSize) {
	rml::Vector3* devPosition;
	float* devRadius;
	int* devGenerateCollisionPointsCount;
	CollisionPoint* devCollisionPoints;

	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devRadius, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devGenerateCollisionPointsCount, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devCollisionPoints, cpSize * sizeof(CollisionPoint));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy pos failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(devRadius, radius, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy minaabb failed!\n");
		goto Error;
	}
	hipMemset(devGenerateCollisionPointsCount, 0, size * sizeof(int));
	hipMemset(devCollisionPoints, 0, cpSize * sizeof(CollisionPoint));

	//CallTheKernel
	// Launch a kernel on the GPU with one thread for each element.
	DetectSphereCollisionsKernel << <numBlocks, threadsPerBlock >> > (devPosition, devRadius, size, devGenerateCollisionPointsCount, devCollisionPoints, cpSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(generateCollisonPointsCount, devGenerateCollisionPointsCount, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy genC failed!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(collisionPoints, devCollisionPoints, cpSize * sizeof(CollisionPoint), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy cp failed!\n");
		goto Error;
	}

Error:
	hipFree(devPosition);
	hipFree(devRadius);
	hipFree(devGenerateCollisionPointsCount);
	hipFree(devCollisionPoints);

	return cudaStatus;
}

hipError_t GPU_HandleBoxCollisions(rml::Vector3* position, rml::Vector3* velocity, rml::Vector3* minAABB, rml::Vector3* maxAABB, int size, float collisionRestitution) {
	rml::Vector3* devPosition;
	rml::Vector3* devVelocity;
	rml::Vector3* devMinAABB;
	rml::Vector3* devMaxAABB;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMinAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMaxAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMinAABB, minAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMaxAABB, maxAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	for (int i = 0; i < size; i++) {
		HandleBoxCollisionsKernel << <numBlocks, threadsPerBlock >> > (devPosition, devVelocity, devMinAABB, devMaxAABB, size, collisionRestitution, i);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	}





	cudaStatus = hipMemcpy(position, devPosition, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devVelocity);
	hipFree(devPosition);
	hipFree(devMinAABB);
	hipFree(devMaxAABB);

	return cudaStatus;
}


hipError_t UpdateBoxes(rml::Vector3* position, rml::Vector3* velocity, rml::Vector3* minAABB, rml::Vector3* maxAABB, int size, float gravity, float dragCoefficient, rml::Vector3 boxMinBound, rml::Vector3 boxMaxBound, float deltaTime) {

	rml::Vector3* devPosition;
	rml::Vector3* devVelocity;
	rml::Vector3* devMinAABB;
	rml::Vector3* devMaxAABB;
	hipError_t cudaStatus;

	dim3 threadsPerBlock(256);
	dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//gpu buffers
	cudaStatus = hipMalloc((void**)&devPosition, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devVelocity, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMinAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMaxAABB, size * sizeof(rml::Vector3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;

	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(devPosition, position, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devVelocity, velocity, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMinAABB, minAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(devMaxAABB, maxAABB, size * sizeof(rml::Vector3), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	UpdateBoxesKernel << <numBlocks, threadsPerBlock >> > (devPosition, devVelocity, devMinAABB, devMaxAABB, gravity, dragCoefficient, boxMinBound, boxMaxBound, deltaTime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(position, devPosition, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(velocity, devVelocity, size * sizeof(rml::Vector3), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devPosition);
	hipFree(devVelocity);
	hipFree(devMinAABB);
	hipFree(devMaxAABB);

	return cudaStatus;
}
